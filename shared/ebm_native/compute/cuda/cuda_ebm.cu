﻿
#include "hip/hip_runtime.h"


__global__ class Addme {

};

__global__ void TestGpuAdd(const int * const pVal1, const int * const pVal2, int * const pResult) {
   Addme();
   const size_t iGpuThread = threadIdx.x;
   pResult[iGpuThread] = pVal1[iGpuThread] + pVal2[iGpuThread];
}

constexpr size_t k_cItems = 5;

bool TestCuda() {
   bool bExitError = true;

   const int aVal1[k_cItems] = { 5, 4, 3, 2, 1 };
   const int aVal2[k_cItems] = { 100, 200, 300, 400, 500 };
   int aResult[k_cItems];
   memset(aResult, 0, sizeof(aResult));

   int * aDeviceVal1 = nullptr;
   int * aDeviceVal2 = nullptr;
   int * aDeviceResult = nullptr;
   hipError_t error;

   error = hipSetDevice(0);
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceVal1, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceVal2, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceResult, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aDeviceVal1, aVal1, k_cItems * sizeof(int), hipMemcpyHostToDevice);
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aDeviceVal2, aVal2, k_cItems * sizeof(int), hipMemcpyHostToDevice);
   if(hipSuccess != error) {
      goto exit_error;
   }

   TestGpuAdd<<<1, k_cItems>>>(aDeviceVal1, aDeviceVal2, aDeviceResult);

   error = hipGetLastError();
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipDeviceSynchronize();
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aResult, aDeviceResult, k_cItems * sizeof(int), hipMemcpyDeviceToHost);
   if(hipSuccess != error) {
      goto exit_error;
   }

   bExitError = false;

exit_error:

   bool bExitHard = false;

   if(nullptr != aDeviceResult) {
      error = hipFree(aDeviceResult);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(nullptr != aDeviceVal2) {
      error = hipFree(aDeviceVal2);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(nullptr != aDeviceVal1) {
      error = hipFree(aDeviceVal1);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(bExitHard) {
      bExitError = true;

      // not much to do with the error if we fail cudaDeviceReset after failing cudaFree
      error = hipDeviceReset();
   }

   return bExitError;
}

