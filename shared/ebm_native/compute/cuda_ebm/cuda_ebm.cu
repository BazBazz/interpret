﻿
#include "hip/hip_runtime.h"


#include <type_traits>

struct TestLoss {
   float m_multiple;

   TestLoss(float multiple) {
      m_multiple = multiple;
   }

   __device__ float CalculateGradient(float target, float prediction) {
      return target * m_multiple + prediction * 100;
   }

   // if the loss function doesn't have a second derivative, then delete the CalculateHessian function.
   __device__ float CalculateHessian(float target, float prediction) {
      return target * 10 + prediction * 100;
   }
};

template <typename TLoss>
__global__ void TestGpuAdd(TLoss * pLoss, const int * const pVal1, const int * const pVal2, int * const pResult) {
   const size_t iGpuThread = threadIdx.x;
   pResult[iGpuThread] = pLoss->CalculateGradient(pVal1[iGpuThread], pVal2[iGpuThread]);
}

constexpr size_t k_cItems = 5;

bool TestCuda() {
   bool bExitError = true;

   const int aVal1[k_cItems] = { 5, 4, 3, 2, 1 };
   const int aVal2[k_cItems] = { 100, 200, 300, 400, 500 };
   int aResult[k_cItems];
   memset(aResult, 0, sizeof(aResult));

   // TODO: unfortunately, I think this means our Loss classes need to be standard_layout and trivially copyable
   // which means no virtual function  :(.  We can use function pointers instead though, even though that's kind
   // of uggly, but at least those will be hidden from the Loss class writer.  In Registration.hpp after
   // calling new TRegistrable... we still have the specific loss type after that call, so we can take a pointer
   // to a function that we inject via the loss MACRO.  Dirty, but it'll get the job done.

   static_assert(std::is_standard_layout<TestLoss>::value,
      "Our Loss type must be a standard layout struct to be inserted into the GPU");
   static_assert(std::is_trivially_copyable<TestLoss>::value,
      "Our Loss type must be a trivial struct to be inserted into the GPU");

   TestLoss loss(9);

   int * aDeviceVal1 = nullptr;
   int * aDeviceVal2 = nullptr;
   int * aDeviceResult = nullptr;
   TestLoss * pDeviceLoss = nullptr;
   hipError_t error;

   error = hipSetDevice(0);
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceVal1, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceVal2, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&aDeviceResult, k_cItems * sizeof(int));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMalloc((void **)&pDeviceLoss, sizeof(TestLoss));
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aDeviceVal1, aVal1, k_cItems * sizeof(int), hipMemcpyHostToDevice);
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aDeviceVal2, aVal2, k_cItems * sizeof(int), hipMemcpyHostToDevice);
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(pDeviceLoss, &loss, sizeof(TestLoss), hipMemcpyHostToDevice);
   if(hipSuccess != error) {
      goto exit_error;
   }

   TestGpuAdd<TestLoss><<<1, k_cItems>>>(pDeviceLoss, aDeviceVal1, aDeviceVal2, aDeviceResult);

   error = hipGetLastError();
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipDeviceSynchronize();
   if(hipSuccess != error) {
      goto exit_error;
   }

   error = hipMemcpy(aResult, aDeviceResult, k_cItems * sizeof(int), hipMemcpyDeviceToHost);
   if(hipSuccess != error) {
      goto exit_error;
   }

   bExitError = false;

exit_error:

   bool bExitHard = false;

   if(nullptr != pDeviceLoss) {
      error = hipFree(pDeviceLoss);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(nullptr != aDeviceResult) {
      error = hipFree(aDeviceResult);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(nullptr != aDeviceVal2) {
      error = hipFree(aDeviceVal2);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(nullptr != aDeviceVal1) {
      error = hipFree(aDeviceVal1);
      if(hipSuccess != error) {
         bExitHard = true;
      }
   }

   if(bExitHard) {
      bExitError = true;

      // not much to do with the error if we fail cudaDeviceReset after failing cudaFree
      error = hipDeviceReset();
   }

   return bExitError;
}

